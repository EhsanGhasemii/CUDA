#include "hip/hip_runtime.h"
#include "../include/cuda_main.h"
#include "../include/gpuerrors.h"
#include "../include/gputimer.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <cstring>





// define our functions ======================================
void fill(float* data, int size);
dim3 getDimGrid(const int n); 
dim3 getDimBlock(const int n); 
__global__ void kernelFunc(double* X_reald, 
						   double* X_imagd,
						   double* R_reald,
						   double* R_imagd,
						   double* Ss_reald,
						   double* Ss_imagd,
						   double* s_reald,
						   double* s_imagd,
						   double* alpha_reald,
						   double* rho_reald, 
						   double* rho_imagd,
						   double* output_reald,
						   double* output_imagd, 

						   int X_size, 
						   int N, 
						   double* testd
						   );
__global__ void matrixInversion(double* inputd,
								double* outputd, 
								const int n, 
								const int m
								);	

__global__ void complexMatrixInversion(double* input_reald,		// input data is "inputd"
									   double* input_imagd,
									   //double* output_reald,    // output data is "outputd"
									   //double* output_imagd,
							           const int n,				// number of matrices is n
									   const int m,		     	// size of each matrix is m*m

									   double* y_n_reald, 
									   double* y_n_imagd, 
									   double* s_reald, 
									   double* s_imagd, 
									   double* rho_reald, 
									   double* rho_imagd, 
									   double* W_reald, 
									   double* W_imagd, 
									   double* W_reald_shr2, 
									   double* W_imagd_shr2, 
									   double* W_reald_shr, 
									   double* W_imagd_shr
									   );			     		// we suppose input data is squre matrix


__global__ void kernelFunc3(double* y_n_reald, 
							double* y_n_imagd,
							double* CRinv_reald,	
							double* CRinv_imagd,
							double* s_reald,   
							double* s_imagd,
							double* rho_reald, 
							double* rho_imagd, 
							double* W_reald, 
							double* W_imagd,
							const int n,			
							const int m		     	
							);		     		

void print_matrix(char* name, double* data, int size, int d_shift);
// ==========================================================


// main body
void gpuKernel(double* y_n_real,
			   double* y_n_imag,
			   double* X_real, 
			   double* X_imag, 
			   double* R_real, 
			   double* R_imag, 
			   double* Ss_real, 
			   double* Ss_imag, 
			   double* s_real, 
			   double* s_imag,
			   double* alpha_real,
			   double* output_real,
			   double* output_imag,
			   double* test, 

			   int data_num, 
			   int y_n_size, 
			   int X_size, 
			   int R_row, 
			   int Ss_size, 
			   int s_size
			   ) {
	
	// print name of device
	struct hipDeviceProp_t p;
    hipGetDeviceProperties(&p, 0);
    printf("Device Name: %s\n", p.name);


	// define our variabels
	int N = 13; 
	int print_flag = 1; 

	// allocate memory in CPU for calculation

	// define our needed variables in GPU
	double* y_n_reald; 
	double* y_n_imagd; 
	double* X_reald;
	double* X_imagd; 
	double* R_reald;
	double* R_imagd; 
	double* Ss_reald; 
	double* Ss_imagd; 
	double* s_reald; 
	double* s_imagd;
	double* alpha_reald;

	double* rho_reald; 
	double* rho_imagd; 
	double* output_reald;
	double* output_imagd;
	double* W_reald;
	double* W_imagd;
	double* W_reald_shr2; 
	double* W_imagd_shr2; 
	double* W_reald_shr; 
	double* W_imagd_shr; 
	double* testd; 


	// allocation memory in GPU
	HANDLE_ERROR(hipMalloc((void**)&y_n_reald, data_num * y_n_size * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&y_n_imagd, data_num * y_n_size * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&X_reald, data_num * X_size * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&X_imagd, data_num * X_size * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&R_reald, R_row * R_row * sizeof(double))); 
	HANDLE_ERROR(hipMalloc((void**)&R_imagd, R_row * R_row * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&Ss_reald, Ss_size * R_row * R_row * sizeof(double))); 
	HANDLE_ERROR(hipMalloc((void**)&Ss_imagd, Ss_size * R_row * R_row * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&s_reald, s_size * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&s_imagd, s_size * sizeof(double))); 
	HANDLE_ERROR(hipMalloc((void**)&alpha_reald, 1 * sizeof(double))); // hard_code: 1 - ch..

	HANDLE_ERROR(hipMalloc((void**)&rho_reald, data_num * X_size * sizeof(double))); // hard_code: 262 - ch..
	HANDLE_ERROR(hipMalloc((void**)&rho_imagd, data_num * X_size * sizeof(double))); // hard_code: 262 - ch..
	HANDLE_ERROR(hipMalloc((void**)&output_reald, data_num * X_size * R_row * R_row * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&output_imagd, data_num * X_size * R_row * R_row * sizeof(double))); 
	HANDLE_ERROR(hipMalloc((void**)&W_reald, data_num * X_size * sizeof(double))); // hard_code: 13 - ch..
	HANDLE_ERROR(hipMalloc((void**)&W_imagd, data_num * X_size * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&W_reald_shr2, data_num * X_size * R_row * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&W_imagd_shr2, data_num * X_size * R_row * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&W_reald_shr, data_num * X_size * R_row * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&W_imagd_shr, data_num * X_size * R_row * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&testd, data_num * X_size * R_row * R_row * sizeof(double))); 

	// copy array from CPU to GPU
	HANDLE_ERROR(hipMemcpy(y_n_reald, y_n_real, data_num * y_n_size * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(y_n_imagd, y_n_imag, data_num * y_n_size * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(X_reald, X_real, data_num * X_size * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(X_imagd, X_imag, data_num * X_size * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(R_reald, R_real, R_row * R_row * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(R_imagd, R_imag, R_row * R_row * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(Ss_reald, Ss_real, Ss_size * R_row * R_row * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(Ss_imagd, Ss_imag, Ss_size * R_row * R_row * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(s_reald, s_real, s_size * sizeof(double), hipMemcpyHostToDevice)); 
	HANDLE_ERROR(hipMemcpy(s_imagd, s_imag, s_size * sizeof(double), hipMemcpyHostToDevice)); 
	HANDLE_ERROR(hipMemcpy(alpha_reald, alpha_real, 1 * sizeof(double), hipMemcpyHostToDevice)); // hard_code: 1 - ch..

	
	// define our threads and blocks dimension
	dim3 dimGrid = getDimGrid(data_num * X_size);
	dim3 dimBlock = getDimBlock(1);

	// transfer processing in CUDA
	double gpu_kernel_time = 0.0;
	GpuTimer timer;
    timer.Start();
	// APC algorithm part1 and part2
	kernelFunc<<< dimGrid,dimBlock >>>(X_reald, 
									   X_imagd, 
									   R_reald, 
									   R_imagd, 
									   Ss_reald, 
									   Ss_imagd, 
									   s_reald, 
									   s_imagd, 
									   alpha_reald, 
									   rho_reald, 
									   rho_imagd, 
									   output_reald, 
									   output_imagd, 

									   X_size, 
									   N, 
									   testd
									   );
	timer.Stop();
	gpu_kernel_time = timer.Elapsed();


	// APC algorithm part3
	dimGrid = getDimGrid(data_num * X_size); 
	dimBlock = getDimBlock(R_row * R_row); 
	complexMatrixInversion<<< dimGrid, dimBlock >>>(output_reald,
													output_imagd,  
													1, 
													13,

													y_n_reald, 
													y_n_imagd, 
													s_reald, 
													s_imagd, 
													rho_reald, 
													rho_imagd, 
													W_reald, 
													W_imagd, 
													W_reald_shr2, 
													W_imagd_shr2, 
													W_reald_shr, 
													W_imagd_shr
													);
													

	// modifying ====================================
	HANDLE_ERROR(hipMemcpy(test, output_reald, data_num * X_size * R_row * R_row * sizeof(double), hipMemcpyDeviceToHost));
	// ==============================================

	// copy result from GPU to CPU memory
	HANDLE_ERROR(hipMemcpy(output_real, W_reald, data_num * X_size * sizeof(double), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(output_imag, W_imagd, data_num * X_size * sizeof(double), hipMemcpyDeviceToHost)); 
	

	// print report
	if (print_flag) {
		std::cout << "this is output of GPU: " << std::endl; 
		char name[20] = "a"; 
		//print_matrix(name, a		 , n);

		strcpy(name, "output_real"); 
		print_matrix(name, output_real, X_size, 0 * X_size); 

		strcpy(name, "output_imag");
		print_matrix(name, output_imag, X_size, 0 * X_size); 

		strcpy(name, "test");
		print_matrix(name, test, 13, 1 * 13 * 13); 

		/*strcpy(name, "inv_real");
		print_matrix(name, inv_real, mat_size * mat_size, 0); 

		strcpy(name, "inv_imag");
		print_matrix(name, inv_imag, mat_size * mat_size, 0); */
		
	}




	// complex matrix inversion =========================================================================
	/*double* input_real; 
	double* input_imag;

	double* inv_real;
	double* inv_imag;

	int mat_size = 3; 

	input_real = (double*)malloc(mat_size * mat_size * sizeof(double)); 
	input_imag = (double*)malloc(mat_size * mat_size * sizeof(double));

	inv_real = (double*)malloc(mat_size * mat_size * sizeof(double));
	inv_imag = (double*)malloc(mat_size * mat_sRize * sizeof(double));

	input_real[0] = 1; 
	input_real[1] = 1; 
	input_real[2] = 2; 
	input_real[3] = 1; 
	input_real[4] = 0; 
	input_real[5] = 1; 
	input_real[6] = 0; 
	input_real[7] = 0; 
	input_real[8] = 0;

	input_imag[0] = 1; 
	input_imag[1] = 0; 
	input_imag[2] = 0; 
	input_imag[3] = 1; 
	input_imag[4] = 2; 
	input_imag[5] = 0; 
	input_imag[6] = 0; 
	input_imag[7] = 0; 
	input_imag[8] = 2; 

	double* input_reald;
	double* input_imagd; 

	double* inv_reald; 
	double* inv_imagd;

	HANDLE_ERROR(hipMalloc((void**)&input_reald, mat_size * mat_size * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&input_imagd, mat_size * mat_size * sizeof(double)));

	HANDLE_ERROR(hipMalloc((void**)&inv_reald, mat_size * mat_size * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&inv_imagd, mat_size * mat_size * sizeof(double)));

	HANDLE_ERROR(hipMemcpy(input_reald, input_real, mat_size * mat_size * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(input_imagd, input_imag, mat_size * mat_size * sizeof(double), hipMemcpyHostToDevice));

	dimGrid = getDimGrid(1); 
	dimBlock = getDimBlock(mat_size * mat_size); 

	complexMatrixInversion<<< dimGrid, dimBlock >>>(input_reald, input_imagd, inv_reald, inv_imagd, 1, mat_size); 

	HANDLE_ERROR(hipMemcpy(inv_real, inv_reald, mat_size * mat_size * sizeof(double), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(inv_imag, inv_imagd, mat_size * mat_size * sizeof(double), hipMemcpyDeviceToHost));*/
	// ==================================================================================================



	// remove array in GPU
	HANDLE_ERROR(hipFree(X_reald));
	HANDLE_ERROR(hipFree(X_imagd));
	HANDLE_ERROR(hipFree(R_reald));
	HANDLE_ERROR(hipFree(R_imagd));
	HANDLE_ERROR(hipFree(Ss_reald));
	HANDLE_ERROR(hipFree(Ss_imagd));
	HANDLE_ERROR(hipFree(s_reald));
	HANDLE_ERROR(hipFree(s_imagd));
	HANDLE_ERROR(hipFree(alpha_reald));
	HANDLE_ERROR(hipFree(output_reald));
	HANDLE_ERROR(hipFree(output_imagd));



	// print a report
	std::cout << "I am in gpuKernel .." << std::endl;


}



// 
dim3 getDimGrid(const int n) {
	dim3 dimGrid(n, 1, 1);

	return dimGrid;
}

//
dim3 getDimBlock(const int n) {
	dim3 dimBlock(n, 1, 1);

	return dimBlock;
}

// kernelFunc
__global__ void kernelFunc(double* X_reald, 
						   double* X_imagd,
						   double* R_reald,
						   double* R_imagd,
						   double* Ss_reald,
						   double* Ss_imagd,
						   double* s_reald,
						   double* s_imagd,
						   double* alpha_reald,
						   double* rho_reald, 
						   double* rho_imagd,
						   double* output_reald,
						   double* output_imagd, 

						   int X_size, 
						   int N, 
						   double* testd
						   ) {
	

	// define our variables
	// !!!!! Please consider that if your input data will be big 
	// you may need to use long format of variables !!!!!!!!!!!!
	//__shared__ double rho_reald_shr[262]; // hard_code: 262 - change it in future
	//__shared__ double rho_imagd_shr[262]; // hard_code: 262 - change it in future

	//__shared__ float Ss_reald_shr[25 * 13 * 13]; 
	//__shared__ float Ss_imagd_shr[25 * 13 * 13]; 

	//double C_reald[262 * 13 * 13]; 
	//double C_imagd[262 * 13 * 13]; 

	// define index of each thread
	long long i;
	i = (blockIdx.z * gridDim.y * gridDim.x) + (blockIdx.y * gridDim.x) + (blockIdx.x);
	i *= blockDim.z * blockDim.y * blockDim.x;
	i += (threadIdx.z * blockDim.y * blockDim.x) + (threadIdx.y * blockDim.x) + (threadIdx.x);

	int block_ind = i % X_size; 


	// !!!!!!!!!!!!!!!!!!!!!!!! this will be updated for multiple chunked data !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!1
	// create rho varible in shared memory
	if (block_ind <= X_size) {               // hard_code: 262 - change it in future
		double my_angle = atan2(X_imagd[i], X_reald[i]); 
		double my_radius = sqrt(X_imagd[i] * X_imagd[i] + X_reald[i] * X_reald[i]); 
		my_radius = pow(my_radius, alpha_reald[0]); 
		my_angle *= alpha_reald[0]; 

		rho_reald[i] = my_radius * cos(my_angle); 
		rho_imagd[i] = my_radius * sin(my_angle); 
	}


	// transfer Ss matrices in shared memory
	/*if (i < 13 * 13) {
		for (int count1 = 0; count1 < 25; ++count1) {
			Ss_reald_shr[count1 * 169 + i] = Ss_reald[count1 * 169 + i]; 
			Ss_imagd_shr[count1 * 169 + i] = Ss_imagd[count1 * 169 + i]; 
		}
	}*/
	
	// wait till all t172.17.0.0/16he data is ready
	__syncthreads(); 


	if ((blockIdx.x <= 261 - 24) || (blockIdx.x <= 261 + 261 - 24 && blockIdx.x >=261)) {     // blockIdx.x <= X_size - 2 * N + 2
		// first part of the algorithm: 25 * (matrix multilplication and addition)
		for (int count1 = 0; count1 < 25; ++count1) {
			for (int count2 = 0; count2 < 13; ++count2) {
				for (int count3 = 0; count3 < 13; ++count3) {         // hard_code: (i-12) in both lines
					output_reald[blockIdx.x * 169 + count2 * 13 + count3] += (rho_reald[blockIdx.x + count1] * Ss_reald[count1 * 169 + count2 * 13 + count3]
																		    - rho_imagd[blockIdx.x + count1] * Ss_imagd[count1 * 169 + count2 * 13 + count3]); 
					output_imagd[blockIdx.x * 169 + count2 * 13 + count3] += (rho_reald[blockIdx.x + count1] * Ss_imagd[count1 * 169 + count2 * 13 + count3]
																		    + rho_imagd[blockIdx.x + count1] * Ss_reald[count1 * 169 + count2 * 13 + count3]);
				}
			}
		}

		// second part of the algorithm: C += R
		for (int count2 = 0; count2 < 13; ++count2) {
			for (int count3 = 0; count3 < 13; ++count3) {
				output_reald[blockIdx.x * 169 + count2 * 13 + count3] += R_reald[count2 * 13 + count3]; 
				output_imagd[blockIdx.x * 169 + count2 * 13 + count3] += R_imagd[count2 * 13 + count3];
			}
		}


		// save rho to use it in next steps
		//rho_reald[i] = rho_reald_shr[i];
		//rho_imagd[i] = rho_imagd_shr[i]; 

	}


}

__global__ void complexMatrixInversion(double* input_reald,		// input data is "inputd"
									   double* input_imagd,
							           const int n,			    // number of matrices is n
									   const int m,		     	// size of each matrix is m*m

									   double* y_n_reald, 
									   double* y_n_imagd, 
									   double* s_reald, 
									   double* s_imagd, 
									   double* rho_reald, 
									   double* rho_imagd, 
									   double* W_reald, 
									   double* W_imagd, 
									   double* W_reald_shr2, 
									   double* W_imagd_shr2, 
									   double* W_reald_shr, 
									   double* W_imagd_shr


									   ) {			     		// we suppose input data is squre matrix


	// define our variables
	__shared__ double out_real[30 * 30];
	__shared__ double out_imag[30 * 30];

	__shared__ double out_real_shr[30 * 30]; 
	__shared__ double out_imag_shr[30 * 30]; 

	__shared__ double in_real[30 * 30]; 
	__shared__ double in_imag[30 * 30]; 
	

	// define index of each thread
	long long i;
	i = (blockIdx.z * gridDim.y * gridDim.x) + (blockIdx.y * gridDim.x) + (blockIdx.x);
	i *= blockDim.z * blockDim.y * blockDim.x;
	i += (threadIdx.z * blockDim.y * blockDim.x) + (threadIdx.y * blockDim.x) + (threadIdx.x);


	// data and thread location
	int mat_num = i / (m * m); 
	int mat_ind = i % (m * m); 
	int mat_row = (i % (m * m)) / m; 
	int mat_col = (i % (m * m)) % m; 





	if (threadIdx.x < 169) {          // i < 169 

		// transfer input data to shared memory
		in_real[mat_ind] = input_reald[i]; 
		in_imag[mat_ind] = input_imagd[i]; 


		// creating eye matrix for gauss jordan elimination
		if (mat_row == mat_col) {	
			out_real[mat_ind] = 1.0; 
			out_imag[mat_ind] = 0.0; 
		}
		else {
			out_real[mat_ind] = 0.0; 
			out_imag[mat_ind] = 0.0; 
		}


		


		// Matrix inversion algorithm main body ======================================== 
		// we use Gauss Jordan Algorithm
		// algorithm: part1 - make the input data upper-triangular
		for (int count1 = 0; count1 < m - 1; ++count1) {
			
			// change current row when its pivot is zero
			if ((input_reald[blockIdx.x * 169 + count1 * m + count1] == 0) && (input_imagd[blockIdx.x * 169 + count1 * m + count1] == 0)) {
				int count2 = count1 + 1; 
				while ((input_reald[blockIdx.x * 169 + count2 * m + count1] == 0) && (input_imagd[blockIdx.x * 169 + count2 * m + count1] == 0) && (count2 < m)) {
					++count2;
				}
				if(mat_row == count1) {
					input_reald[blockIdx.x * 169 + mat_ind] += input_reald[blockIdx.x * 169 + count2 * m + mat_col]; // ch ..
					input_imagd[blockIdx.x * 169 + mat_ind] += input_imagd[blockIdx.x * 169 + count2 * m + mat_col]; // ch ..

					out_real[mat_ind] += out_real[count2 * m + mat_col]; 
					out_imag[mat_ind] += out_imag[count2 * m + mat_col]; 
				}
				__syncthreads(); 	
			}

			if (mat_row > count1) {
				double mul_real = in_real[mat_row * m + count1] * in_real[count1 * m + count1]
								+ in_imag[mat_row * m + count1] * in_imag[count1 * m + count1];
				mul_real /= (in_real[count1 * m + count1] * in_real[count1 * m + count1]
						   + in_imag[count1 * m + count1] * in_imag[count1 * m + count1]); 

				double mul_imag = in_imag[mat_row * m + count1] * in_real[count1 * m + count1]
								- in_real[mat_row * m + count1] * in_imag[count1 * m + count1];
				mul_imag /= (in_real[count1 * m + count1] * in_real[count1 * m + count1]
						   + in_imag[count1 * m + count1] * in_imag[count1 * m + count1]); 

				in_real[mat_ind] -= (mul_real * in_real[count1 * m + mat_col]
								   - mul_imag * in_imag[count1 * m + mat_col]); 
				in_imag[mat_ind] -= (mul_real * in_imag[count1 * m + mat_col]
								   + mul_imag * in_real[count1 * m + mat_col]);

				out_real[mat_ind] -= (mul_real * out_real[count1 * m + mat_col]
									- mul_imag * out_imag[count1 * m + mat_col]);
				out_imag[mat_ind] -= (mul_real * out_imag[count1 * m + mat_col]
									+ mul_imag * out_real[count1 * m + mat_col]);
			}

			// wait till all the data is changed
			__syncthreads(); 
		}


		// algorithm: part2 - make the input data lower-triangular
		for (int count1 = m - 1; count1 > 0; --count1) {
			if (mat_row < count1) {
				double mul_real = in_real[mat_row * m + count1] * in_real[count1 * m + count1]
								+ in_imag[mat_row * m + count1] * in_imag[count1 * m + count1];
				mul_real /= (in_real[count1 * m + count1] * in_real[count1 * m + count1]
						   + in_imag[count1 * m + count1] * in_imag[count1 * m + count1]); 

				double mul_imag = in_imag[mat_row * m + count1] * in_real[count1 * m + count1]
								- in_real[mat_row * m + count1] * in_imag[count1 * m + count1];
				mul_imag /= (in_real[count1 * m + count1] * in_real[count1 * m + count1]
						   + in_imag[count1 * m + count1] * in_imag[count1 * m + count1]); 

				in_real[mat_ind] -= (mul_real * in_real[count1 * m + mat_col]
								   - mul_imag * in_imag[count1 * m + mat_col]); 
				in_imag[mat_ind] -= (mul_real * in_imag[count1 * m + mat_col]
								   + mul_imag * in_real[count1 * m + mat_col]);

				out_real[mat_ind] -= (mul_real * out_real[count1 * m + mat_col]
									- mul_imag * out_imag[count1 * m + mat_col]);
				out_imag[mat_ind] -= (mul_real * out_imag[count1 * m + mat_col]
									+ mul_imag * out_real[count1 * m + mat_col]);
			}

			// wait till all the data is changed
			__syncthreads(); 
		}

		// algorithm: part3 - normalize input data to create matrix inversion
		out_real_shr[mat_ind] = (out_real[mat_ind] * in_real[mat_row * m + mat_row]
							   + out_imag[mat_ind] * in_imag[mat_row * m + mat_row])
							   / (in_real[mat_row * m + mat_row] * in_real[mat_row * m + mat_row]
								+ in_imag[mat_row * m + mat_row] * in_imag[mat_row * m + mat_row]);
		
		out_imag_shr[mat_ind] = (out_imag[mat_ind] * in_real[mat_row * m + mat_row]
							   - out_real[mat_ind] * in_imag[mat_row * m + mat_row])
								/ (in_real[mat_row * m + mat_row] * in_real[mat_row * m + mat_row]
								 + in_imag[mat_row * m + mat_row] * in_imag[mat_row * m + mat_row]);

	} // if (i < 169)



	// ============================================================================

	// define our variables
/*	__shared__ double W_reald_shr[25]; 
	__shared__ double W_imagd_shr[25]; 
	__shared__ double W_reald_shr2[25]; 
	__shared__ double W_imagd_shr2[25];*/


	// =======================================
	//double W_reald_shr[30 * 13]; // ..........
	//double W_imagd_shr[30 * 13]; // ..........
	//double W_reald_shr2[30 * 13]; // ch ......
	//double W_imagd_shr2[30 * 13]; // ch .......
	// =======================================

	mat_num = 0; 


	// initialize shared memroy to zero
	if (threadIdx.x < 13) {        // i < 13
		W_reald_shr[blockIdx.x * 13 + threadIdx.x] = 0.0; // ch ...................
		W_imagd_shr[blockIdx.x * 13 + threadIdx.x] = 0.0; // ch ...................
	}

	__syncthreads(); 

	// APC algorithm pqrt4: inv(C+R) * s 
	if (threadIdx.x < 13) {            // hard_code: 13 - ch..
		for (int count1 = 0; count1 < 13; ++count1) {     // count1 < 13
			W_reald_shr[blockIdx.x * 13 + threadIdx.x] += out_real_shr[threadIdx.x * 13 + count1] * s_reald[count1]; 
			W_reald_shr[blockIdx.x * 13 + threadIdx.x] -= out_imag_shr[threadIdx.x * 13 + count1] * s_imagd[count1]; 

			W_imagd_shr[blockIdx.x * 13 + threadIdx.x] += out_real_shr[threadIdx.x * 13 + count1] * s_imagd[count1]; 
			W_imagd_shr[blockIdx.x * 13 + threadIdx.x] += out_imag_shr[threadIdx.x * 13 + count1] * s_reald[count1]; 
		}

	}

	__syncthreads(); 
	
	// APC algorithm part5: W = inv(C+R) * s * rho
	if (threadIdx.x < 13) {
		W_reald_shr2[blockIdx.x * 13 + threadIdx.x] = W_reald_shr[blockIdx.x * 13 + threadIdx.x] * rho_reald[blockIdx.x + 12]
									   			    - W_imagd_shr[blockIdx.x * 13 + threadIdx.x] * rho_imagd[blockIdx.x + 12];        // handle this for data_num > 1 
		W_imagd_shr2[blockIdx.x * 13 + threadIdx.x] = W_reald_shr[blockIdx.x * 13 + threadIdx.x] * rho_imagd[blockIdx.x + 12]
											  	    + W_imagd_shr[blockIdx.x * 13 + threadIdx.x] * rho_reald[blockIdx.x + 12];
	}

	__syncthreads(); 

	// modifying ====================================
	/*if (threadIdx.x < 13 && blockIdx.x == 1) {
		W_reald[threadIdx.x] = input_reald[i]; 
		W_imagd[threadIdx.x] = in_real[mat_ind]; 
	}*/
	// ==============================================


	// APC algorithm part6: W.t() * y_noisy
	if (threadIdx.x == 0) {			// hard_code: 13 - ch..

		W_reald[blockIdx.x] = 0.0; // 0.0
		W_imagd[blockIdx.x] = 0.0; // 0.0

		for (int count1 = 0; count1 < 13; ++count1) {        // count1 < 13 
			W_reald[blockIdx.x] += W_reald_shr2[blockIdx.x * 13 + count1] * y_n_reald[blockIdx.x + 12 + count1]
						 	     - W_imagd_shr2[blockIdx.x * 13 + count1] * y_n_imagd[blockIdx.x + 12 + count1];
			W_imagd[blockIdx.x] += W_reald_shr2[blockIdx.x * 13 + count1] * y_n_imagd[blockIdx.x + 12 + count1]
							     + W_imagd_shr2[blockIdx.x * 13 + count1] * y_n_reald[blockIdx.x + 12 + count1]; 
		}
	}

}





/*
__global__ void matrixInversion(double* inputd,		// input data is "inputd"
								double* outputd,	// output data is "outputd"
								const int n,				// number of matrices is n
								const int m				// size of each matrix is m*m
								) {					// we suppose input data is squre matrix




	// define our variables
	__shared__ double out[3 * 3];
	__shared__ double in[3 * 3]; 
	

	// define index of each thread
	long long i;
	i = (blockIdx.z * gridDim.y * gridDim.x) + (blockIdx.y * gridDim.x) + (blockIdx.x);
	i *= blockDim.z * blockDim.y * blockDim.x;
	i += (threadIdx.z * blockDim.y * blockDim.x) + (threadIdx.y * blockDim.x) + (threadIdx.x);


	// data and thread location
	int mat_num = i / (m * m); 
	int mat_ind = i % (m * m); 
	int mat_row = (i % (m * m)) / m; 
	int mat_col = (i % (m * m)) % m; 

	// transfer input data to shared memory
	in[mat_ind] = inputd[i]; 


	// creating eye matrix for gauss jordan elimination
	if (mat_row == mat_col) {	
		out[mat_ind] = 1.0; 
	}
	else {
		out[mat_ind] = 0.0; 
	}

	// Matrix inversion algorithm main body ======================================== 
	// we use Gauss Jordan Algorithm
	// algorithm: part1 - make the input data upper-triangular
	for (int count1 = 0; count1 < m - 1; ++count1) {
		
		// change current row when its pivot is zero
		if (in[count1 * m + count1] == 0) {
			int count2 = count1 + 1; 
			while ((in[count2 * m + count1] == 0) && (count2 < m)) {
				++count2;
			}
			if(mat_row == count1) {
				in[mat_ind] += in[count2 * m + mat_col];
				out[mat_ind] += out[count2 * m + mat_col]; 
			}
			__syncthreads(); 	
		}


		if (mat_row > count1) {
			double mul = in[mat_row * m + count1] / in[count1 * m + count1]; 
			in[mat_ind] -= mul * in[count1 * m + mat_col];
			out[mat_ind] -= mul * out[count1 * m + mat_col];
		}

		// wait till all the data is changed
		__syncthreads(); 
	}


	// algorithm: part2 - make the input data lower-triangular
	for (int count1 = m - 1; count1 > 0; --count1) {
		if (mat_row < count1) {
			double mul = in[mat_row * m + count1] / in[count1 * m + count1]; 
			in[mat_ind] -= mul * in[count1 * m + mat_col];
			out[mat_ind] -= mul * out[count1 * m + mat_col];
		}

		// wait till all the data is changed
		__syncthreads(); 
	}

	// algorithm: part3 - normalize input data to create matrix inversion
	out[mat_ind] /= in[mat_row * m + mat_row]; 
	// ============================================================================


	outputd[i] = out[mat_ind]; 
}
*/






// print matrix
void print_matrix(char* name, double* data, int size, int d_shift) {
	printf("arr : %s\n", name);
	for (int i=0+d_shift; i<size+d_shift; ++i) {
		printf("%d : %f\n", i, data[i]); 
	}
	printf("--------------------\n"); 
}
