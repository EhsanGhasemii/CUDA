#include "hip/hip_runtime.h"
#include "../include/cuda_main.h"
#include "../include/gpuerrors.h"
#include "../include/gputimer.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <cstring>




// define our functions ======================================
void fill(float* data, int size);
dim3 getDimGrid(const int n); 
dim3 getDimBlock(const int n); 


__global__ void matrixInversion(double* inputd,
								double* outputd, 
								const int n, 
								const int m
								);	

__global__ void X_to_rho(double* X_reald, 
						 double* X_imagd, 
						 double* rho_reald, 
						 double* rho_imagd, 
						 double* alpha_reald, 
						 int alpha_indx
						 );




__global__ void complexMatrixInversion(double* input_reald,		// input data is "inputd"
									   double* input_imagd,
									   //double* output_reald,    // output data is "outputd"
									   //double* output_imagd,
							           const int n,				// number of matrices is n
									   const int m,		     	// size of each matrix is m*m

									   double* y_n_reald, 
									   double* y_n_imagd, 
									   double* s_reald, 
									   double* s_imagd, 
									   double* rho_reald, 
									   double* rho_imagd, 
									   double* W_reald, 
									   double* W_imagd, 
									   double* W_reald_shr2, 
									   double* W_imagd_shr2, 
									   double* W_reald_shr, 
									   double* W_imagd_shr, 

									   double* out_reald, 
									   double* out_imagd, 
									   double* Ss_reald, 
									   double* Ss_imagd, 
									   double* R_reald,
									   double* R_imagd, 
									   
									   double* test_reald, 
									   double* test_imagd, 
									   int alpha_indx, 
									   int N, 
									   int X_size, 
									   int data_num, 
									   int batch_size
									   );			     		// we suppose input data is squre matrix



void print_matrix(char* name, double* data, int size, int d_shift);
// ==========================================================


// modifying =================
void testGpu(double* y_n_real
			 //double* y_n_imag, 
			 //double* X_real, 
			 //double* X_imag, 
			 //double* rho_real, 
			 //double* rho_imag,
			 //double* R_real, 
			 //double* R_imag, 
			 //double* Ss_real,
			 //double* Ss_imag
		) {

	// print name of device
	//struct hipDeviceProp_t p;
    //hipGetDeviceProperties(&p, 0);
    //printf("Device Name: %s\n", p.name);
	GpuTimer p0; 
	printf("salam..\n");
}



// main body
void gpuKernel(double* y_n_real,
			   double* y_n_imag,
			   double* X_real, 
			   double* X_imag,
		//	   double* rho_real, 
		//	   double* rho_imag, 
			   double* R_real, 
			   double* R_imag, 
			   double* Ss_real, 
			   double* Ss_imag, 
			   double* s_real, 
			   double* s_imag,
			   double* alpha_real,
			   double* output_real,
			   double* output_imag,
			   double* test_real,
			   double* test_imag, 

			   int batch_size, 
			   int data_num, 
			   int y_n_size, 
			   int X_size, 
			   int R_row, 
			   int Ss_size, 
			   int s_size, 
			   int alpha_size
			   ) {




	// print name of device
	struct hipDeviceProp_t p;
    hipGetDeviceProperties(&p, 0);
    printf("Device Name: %s\n", p.name);

	// modifying ====================================================
	GpuTimer p12;
	GpuTimer p23; 
	GpuTimer p34; 
	GpuTimer p45; 
	GpuTimer p15; 
    p12.Start();
	p15.Start(); 

	// ==============================================================

	// define our variabels
	int N = 13; 
	int print_flag = 0; 

	// allocate memory in CPU for calculation

	// define our needed variables in GPU
	double* y_n_reald; 
	double* y_n_imagd; 
	double* X_reald;
	double* X_imagd; 
	double* R_reald;
	double* R_imagd; 
	double* Ss_reald; 
	double* Ss_imagd; 
	double* s_reald; 
	double* s_imagd;
	double* alpha_reald;

	double* rho_reald; 
	double* rho_imagd; 
	double* output_reald;
	double* output_imagd;
	double* W_reald_shr2; 
	double* W_imagd_shr2; 
	double* W_reald_shr; 
	double* W_imagd_shr; 
	//double* testd; 

	double* out_reald; 
	double* out_imagd; 

	double* test_reald; 
	double* test_imagd; 




	// allocation memory in GPU
	HANDLE_ERROR(hipMalloc((void**)&y_n_reald, batch_size * data_num * y_n_size * sizeof(double)));	// size: batch_size * data_num * y_n_size	space: <1MB
	HANDLE_ERROR(hipMalloc((void**)&y_n_imagd, batch_size * data_num * y_n_size * sizeof(double)));	// size: batch_size * data_num * y_n_size	space: <1MB
	HANDLE_ERROR(hipMalloc((void**)&X_reald, batch_size * data_num * X_size * sizeof(double)));		// size: batch_size * data_num * X_size		space: <1MB
	HANDLE_ERROR(hipMalloc((void**)&X_imagd, batch_size * data_num * X_size * sizeof(double)));		// size: batch_size * data_num * X_size		space: <1MB
	HANDLE_ERROR(hipMalloc((void**)&R_reald, R_row * R_row * sizeof(double)));							// size: R_row * R_row						space: <1MB
	HANDLE_ERROR(hipMalloc((void**)&R_imagd, R_row * R_row * sizeof(double)));							// size: R_row * r_row						space: <1MB
	HANDLE_ERROR(hipMalloc((void**)&Ss_reald, Ss_size * R_row * R_row * sizeof(double)));				// size: Ss_size * R_row * R_row			space: <1MB
	HANDLE_ERROR(hipMalloc((void**)&Ss_imagd, Ss_size * R_row * R_row * sizeof(double)));				// size: Ss_size * R_row * R_row			space: <1MB
	HANDLE_ERROR(hipMalloc((void**)&s_reald, s_size * sizeof(double)));								// size: s_size								space: <1MB
	HANDLE_ERROR(hipMalloc((void**)&s_imagd, s_size * sizeof(double)));								// size: s_size								space: <1MB
	HANDLE_ERROR(hipMalloc((void**)&alpha_reald, alpha_size * sizeof(double)));						// size: alpha_size							space: <1MB

	HANDLE_ERROR(hipMalloc((void**)&rho_reald, batch_size * data_num * X_size * sizeof(double)));						// size: batch_size * data_num * X_size
	HANDLE_ERROR(hipMalloc((void**)&rho_imagd, batch_size * data_num * X_size * sizeof(double)));						// size: batch_size * data_num * X_size
	HANDLE_ERROR(hipMalloc((void**)&output_reald, batch_size * data_num * X_size * R_row * R_row * sizeof(double)));	// size: batch_size * data_num * X_size * R_row * R_row		space: 38MB
	HANDLE_ERROR(hipMalloc((void**)&output_imagd, batch_size * data_num * X_size * R_row * R_row * sizeof(double)));	// size: batch_size * data_num * X_size * R_row * R_row		space: 38MB
	HANDLE_ERROR(hipMalloc((void**)&W_reald_shr2, batch_size * data_num * X_size * R_row * sizeof(double)));			// size: batch_size * data_num * X_size * R_row				space: 4MB
	HANDLE_ERROR(hipMalloc((void**)&W_imagd_shr2, batch_size * data_num * X_size * R_row * sizeof(double)));			// size: batch_size * data_num * X_size * R_row				space: 4MB
	HANDLE_ERROR(hipMalloc((void**)&W_reald_shr, batch_size * data_num * X_size * R_row * sizeof(double)));			// size: batch_size * data_nym * X_size * R_row				space: 4MB
	HANDLE_ERROR(hipMalloc((void**)&W_imagd_shr, batch_size * data_num * X_size * R_row * sizeof(double)));			// size: batch_size * data_num * X_size * R_row				space: 4MB
	HANDLE_ERROR(hipMalloc((void**)&out_reald, batch_size * data_num * X_size * R_row * R_row * sizeof(double)));		// size: batch_size * data_num * X_size * R_row * R_row		space: 38MB	
	HANDLE_ERROR(hipMalloc((void**)&out_imagd, batch_size * data_num * X_size * R_row * R_row * sizeof(double)));		// size: batch_size * data_num * X_size * R_row * R_row		space: 38MB

	HANDLE_ERROR(hipMalloc((void**)&test_reald, batch_size * data_num * X_size * R_row * R_row * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&test_imagd, batch_size * data_num * X_size * R_row * R_row * sizeof(double))); 



	// modifying ================================================
	/* lets check speed of our algoritm
	   */
	//for (int count0 = 0; count0 < 256; ++count0) {
	// ==========================================================

	// copy array from CPU to GPU
	HANDLE_ERROR(hipMemcpy(y_n_reald, y_n_real, batch_size * data_num * y_n_size * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(y_n_imagd, y_n_imag, batch_size * data_num * y_n_size * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(X_reald, X_real, batch_size * data_num * X_size * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(X_imagd, X_imag, batch_size * data_num * X_size * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(R_reald, R_real, R_row * R_row * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(R_imagd, R_imag, R_row * R_row * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(Ss_reald, Ss_real, Ss_size * R_row * R_row * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(Ss_imagd, Ss_imag, Ss_size * R_row * R_row * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(s_reald, s_real, s_size * sizeof(double), hipMemcpyHostToDevice)); 
	HANDLE_ERROR(hipMemcpy(s_imagd, s_imag, s_size * sizeof(double), hipMemcpyHostToDevice)); 
	HANDLE_ERROR(hipMemcpy(alpha_reald, alpha_real, alpha_size * sizeof(double), hipMemcpyHostToDevice));

	
	// define our threads and blocks dimension
	p12.Stop(); 
	p23.Start(); 


	dim3 dimGrid; 
	dim3 dimBlock; 

	for (int count1 = 0; count1 < alpha_size; ++count1) {

		// create rho from X
		dimGrid = getDimGrid(batch_size * data_num * X_size); 
		dimBlock = getDimBlock(1); 
		X_to_rho<<< dimGrid, dimBlock >>>(X_reald, 
										  X_imagd, 
										  rho_reald, 
										  rho_imagd, 
										  alpha_reald, 
										  count1
										  ); 
				

		p23.Stop(); 
		p34.Start(); 

		// APC algorithm part3
		dimGrid = getDimGrid(data_num * X_size); 
		dimBlock = getDimBlock(batch_size * R_row); 
		complexMatrixInversion<<< dimGrid, dimBlock >>>(output_reald,
														output_imagd,  
														1, 
														13,

														y_n_reald,		
														y_n_imagd,		
														s_reald,	
														s_imagd,		
														rho_reald,		// depend on alpha step
														rho_imagd,		// depend on alpha step
														X_reald,		// result
														X_imagd,		// result
														W_reald_shr2, 
														W_imagd_shr2, 
														W_reald_shr, 
														W_imagd_shr, 

														out_reald, 
														out_imagd, 
														Ss_reald, 
														Ss_imagd, 
														R_reald, 
														R_imagd, 

														test_reald, 
														test_imagd, 
														count1, 
														N, 
														X_size, 
														data_num, 
														batch_size

														);
		

		p34.Stop(); 
		p45.Start(); 

	} // alpha iterations

	

	// modifying =========================================================
	//}
	// ===================================================================



	// modifying ====================================
	//HANDLE_ERROR(hipMemcpy(test, output_reald, data_num * X_size * R_row * R_row * sizeof(double), hipMemcpyDeviceToHost));
	// ==============================================

	// copy result from GPU to CPU memory
	HANDLE_ERROR(hipMemcpy(output_real, X_reald, batch_size * data_num * X_size * sizeof(double), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(output_imag, X_imagd, batch_size * data_num * X_size * sizeof(double), hipMemcpyDeviceToHost));


	// modifying ====================================
	HANDLE_ERROR(hipMemcpy(test_real, test_reald, batch_size * data_num * X_size * R_row * R_row * sizeof(double), hipMemcpyDeviceToHost)); 
	HANDLE_ERROR(hipMemcpy(test_imag, test_imagd, batch_size * data_num * X_size * R_row * R_row * sizeof(double), hipMemcpyDeviceToHost)); 
	// ==============================================
	

	// print report
	if (print_flag) {
		std::cout << "this is output of GPU: " << std::endl; 
		char name[20] = "a"; 
		//print_matrix(name, a		 , n);

		//strcpy(name, "output_real"); 
		//print_matrix(name, output_real, X_size, data_num * X_size); 

		strcpy(name, "output_imag");
		print_matrix(name, test_real, X_size, 1 * data_num * X_size * 13 + 62 * X_size * 13 + 189 * 13); 
		
	}




	// remove array in GPU
	HANDLE_ERROR(hipFree(X_reald));
	HANDLE_ERROR(hipFree(X_imagd));
	HANDLE_ERROR(hipFree(R_reald));
	HANDLE_ERROR(hipFree(R_imagd));
	HANDLE_ERROR(hipFree(Ss_reald));
	HANDLE_ERROR(hipFree(Ss_imagd));
	HANDLE_ERROR(hipFree(s_reald));
	HANDLE_ERROR(hipFree(s_imagd));
	HANDLE_ERROR(hipFree(alpha_reald));
	HANDLE_ERROR(hipFree(output_reald));
	HANDLE_ERROR(hipFree(output_imagd));

	HANDLE_ERROR(hipFree(y_n_reald));
	HANDLE_ERROR(hipFree(y_n_imagd));
	HANDLE_ERROR(hipFree(rho_reald));
	HANDLE_ERROR(hipFree(rho_imagd));
	HANDLE_ERROR(hipFree(W_reald_shr2));
	HANDLE_ERROR(hipFree(W_imagd_shr2));
	HANDLE_ERROR(hipFree(W_reald_shr));
	HANDLE_ERROR(hipFree(W_imagd_shr));

	HANDLE_ERROR(hipFree(test_reald));
	HANDLE_ERROR(hipFree(out_reald));
	HANDLE_ERROR(hipFree(out_imagd));




	// print a report
	std::cout << "I am in gpuKernel .." << std::endl;

	
	// modifying ====================================
	p15.Stop();
	p45.Stop(); 
	double t12 = 0.0; 
	double t23 = 0.0; 
	double t34 = 0.0; 
	double t45 = 0.0; 
	double t15 = 0.0; 
	printf("=================================================\n");
	t12 = p12.Elapsed();
	t23 = p23.Elapsed(); 
	t34 = p34.Elapsed();
	t45 = p45.Elapsed(); 
	t15 = p15.Elapsed();
	printf("TIME OF GPU PARTS:\nt12: %g\nt23: %g\nt34: %g\nt45: %g\nt15: %g\n", t12, t23, t34, t45, t15); 
	printf("=================================================\n");
	// ==============================================




}



// 
dim3 getDimGrid(const int n) {
	dim3 dimGrid(n, 1, 1);

	return dimGrid;
}

//
dim3 getDimBlock(const int n) {
	dim3 dimBlock(n, 1, 1);

	return dimBlock;
}

__global__ void X_to_rho(double* X_reald, 
						 double* X_imagd, 
						 double* rho_reald, 
						 double* rho_imagd, 
						 double* alpha_reald, 
						 int alpha_indx
						 ){

	double my_angle = atan2(X_imagd[blockIdx.x], X_reald[blockIdx.x]); 
	double my_radius = sqrt(X_imagd[blockIdx.x] * X_imagd[blockIdx.x] + X_reald[blockIdx.x] * X_reald[blockIdx.x]); 
	my_radius = pow(my_radius, alpha_reald[alpha_indx]); 
	my_angle *= alpha_reald[alpha_indx]; 

	rho_reald[blockIdx.x] = my_radius * cos(my_angle); 
	rho_imagd[blockIdx.x] = my_radius * sin(my_angle); 
}


/* How can I improve speed of the algorithm?
   - modify number of threads for inverse matrix algorithm. 
   - use registers for multy writhing in a memory location. 
   - we can not use shared memory. 
   */
__global__ void complexMatrixInversion(double* input_reald,		// input data is "inputd"
									   double* input_imagd,
							           const int n,			    // number of matrices is n
									   const int m,		     	// size of each matrix is m*m

									   double* y_n_reald, 
									   double* y_n_imagd, 
									   double* s_reald, 
									   double* s_imagd, 
									   double* rho_reald, 
									   double* rho_imagd, 
									   double* W_reald, 
									   double* W_imagd, 
									   double* W_reald_shr2, 
									   double* W_imagd_shr2, 
									   double* W_reald_shr, 
									   double* W_imagd_shr, 

									   double* out_reald, 
									   double* out_imagd, 
									   double* Ss_reald, 
									   double* Ss_imagd, 
									   double* R_reald, 
									   double* R_imagd, 
									   
									   double* test_reald, 
									   double* test_imagd, 
									   int alpha_indx, 
									   int N, 
									   int X_size, 
									   int data_num, 
									   int batch_size

									   ) {			     		// we suppose input data is squre matrix



	// =========================================================================
	
//	int thr_row = threadIdx.x / 13; 
	int thr_col = threadIdx.x % 13;
	int thr_batch = threadIdx.x / 13; 

	double out_real_temp; 
	double out_imag_temp;

	double W_real_temp;
	double W_imag_temp; 


//	if ((blockIdx.x % X_size <= X_size - 2 * N + 2)) {     // blockIdx.x <= X_size - 2 * N + 2
		// first part of the algorithm: 25 * (matrix multilplication and addition)

		for (int thr_row = 0; thr_row < 13; ++thr_row) {
			__syncthreads(); // APPP
			input_reald[thr_batch * data_num * X_size * 169 + blockIdx.x * 169 + thr_row * 13 + thr_col] = 0.0; 
			input_imagd[thr_batch * data_num * X_size * 169 + blockIdx.x * 169 + thr_row * 13 + thr_col] = 0.0; 
			__syncthreads();  // APPP
		}
		

		__syncthreads();  // AP

		for (int count1 = 0; count1 < 25; ++count1) {         // 25
			__syncthreads();  // AP
			for (int thr_row = 0; thr_row < 13; ++thr_row) {
				__syncthreads(); // APPP
				input_reald[thr_batch*data_num*X_size*169+blockIdx.x*169+thr_row*13+thr_col] += (rho_reald[thr_batch*data_num*X_size+blockIdx.x+count1]*Ss_reald[count1*169+thr_row*13+thr_col]
																							   - rho_imagd[thr_batch*data_num*X_size+blockIdx.x+count1]*Ss_imagd[count1*169+thr_row*13+thr_col]); 
				__syncthreads();  // AP
				input_imagd[thr_batch*data_num*X_size*169+blockIdx.x*169+thr_row*13+thr_col] += (rho_reald[thr_batch*data_num*X_size+blockIdx.x+count1]*Ss_imagd[count1*169+thr_row*13+thr_col]
																							   + rho_imagd[thr_batch*data_num*X_size+blockIdx.x+count1]*Ss_reald[count1*169+thr_row*13+thr_col]);
				__syncthreads(); // APPP


			}
		__syncthreads();  // AP
		}
	
		__syncthreads();


		// second part of the algorithm: C += R
		for (int thr_row = 0; thr_row < 13; ++thr_row) {
			input_reald[thr_batch * data_num * X_size * 169 + blockIdx.x * 169 + thr_row * 13 + thr_col] += R_reald[thr_row * 13 + thr_col]; 
			__syncthreads(); // APPP
			input_imagd[thr_batch * data_num * X_size * 169 + blockIdx.x * 169 + thr_row * 13 + thr_col] += R_imagd[thr_row * 13 + thr_col];
			__syncthreads(); // APPP
		}



		// modifying ============================
		//test_reald[thr_batch * data_num * X_size * 13 + blockIdx.x * 13 + thr_col] = W_imagd_shr[thr_batch * data_num * X_size * 13 + blockIdx.x * 13 + thr_col];
		//__syncthreads(); 
		// ======================================





//	} // check block

	// ==========================================================================
	// define our variables
	__shared__ double out_real[13 * 13];
	__shared__ double out_imag[13 * 13];

	__shared__ double out_real_shr[13 * 13]; 
	__shared__ double out_imag_shr[13 * 13]; 

	__shared__ double in_real[13 * 13]; 
	__shared__ double in_imag[13 * 13]; 


	// define index of each thread
	long long i;
	i = (blockIdx.z * gridDim.y * gridDim.x) + (blockIdx.y * gridDim.x) + (blockIdx.x);
	i *= blockDim.z * blockDim.y * blockDim.x;
	i += (threadIdx.z * blockDim.y * blockDim.x) + (threadIdx.y * blockDim.x) + (threadIdx.x);


	// data and thread location
	int mat_num = i / (m);					// 0, 1, ... data_num * blablabla
	int mat_ind = i % (m);					// 0, 1, ... 13
	//int mat_row = (i % (m * m)) / m; 
	int mat_col = i % m;					// 0, 1, ... 13




//	if (threadIdx.x < 13) {          // i < 13

		// creating eye matrix for gauss jordan elimination
		for (int mat_row = 0; mat_row < 13; ++mat_row) {
			if (mat_row == mat_col) {	
				out_reald[thr_batch * data_num * X_size * 169 + blockIdx.x * 169 + mat_row * 13 + mat_col] = 1.0; 
				out_imagd[thr_batch * data_num * X_size * 169 + blockIdx.x * 169 + mat_row * 13 + mat_col] = 0.0; 
			}
			else {
				out_reald[thr_batch * data_num * X_size * 169 + blockIdx.x * 169 + mat_row * 13 + mat_col] = 0.0; 
				out_imagd[thr_batch * data_num * X_size * 169 + blockIdx.x * 169 + mat_row * 13 + mat_col] = 0.0; 
			}
			__syncthreads(); // APPP
		}

// Matrix inversion algorithm main body ======================================== 
		// we use Gauss Jordan Algorithm
		// algorithm: part1 - make the input data upper-triangular
		for (int count1 = 0; count1 < m - 1; ++count1) {
		
		__syncthreads();  // AP
			// change current row when its pivot is zero
			if ((input_reald[thr_batch*data_num*X_size*169+blockIdx.x*169+count1*m+count1] == 0) && (input_imagd[thr_batch*data_num*X_size*169+blockIdx.x*169+count1*m+count1] == 0)) {
				int count2 = count1 + 1; 
				while ((input_reald[thr_batch*data_num*X_size*169+blockIdx.x*169+count2*m+count1]==0) && (input_imagd[thr_batch*data_num*X_size*169+blockIdx.x*169+count2*m+count1]==0)&&(count2<m)) {
					++count2;
				}
				//if(mat_row == count1) {
				input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * 13 + mat_col] += input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count2 * m + mat_col]; // ch ..
				input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * 13 + mat_col] += input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count2 * m + mat_col]; // ch ..

				out_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * 13 + mat_col] += out_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count2 * m + mat_col]; 
				out_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * 13 + mat_col] += out_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count2 * m + mat_col]; 
				//}
				__syncthreads(); 	
			}


		__syncthreads();  // AP



			for (int mat_row = count1 + 1; mat_row < 13; ++mat_row) {
				double mul_real = input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row* m + count1] *input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1]
								+ input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row* m + count1] *input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1];
					 mul_real /= (input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1] *input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1]
								+ input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1] *input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1]); 

				double mul_imag = input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row* m + count1] *input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1]
								- input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row* m + count1] *input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1];
					 mul_imag /= (input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1] *input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1]
								+ input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1] *input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1]); 

				input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * 13 + mat_col] -= (mul_real * input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + mat_col]
																									   - mul_imag * input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + mat_col]); 
				input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * 13 + mat_col] -= (mul_real * input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + mat_col]
																									   + mul_imag * input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + mat_col]);
				__syncthreads(); // APPP

				out_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * 13 + mat_col] -= (mul_real * out_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + mat_col]
																									 - mul_imag * out_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + mat_col]);
				out_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * 13 + mat_col] -= (mul_real * out_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + mat_col]
																									 + mul_imag * out_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + mat_col]);
				__syncthreads(); // APPP

			}

			// wait till all the data is changed
			__syncthreads(); 
		}


		// algorithm: part2 - make the input data lower-triangular
		for (int count1 = m - 1; count1 > 0; --count1) {
		

		__syncthreads();  // AP

			for (int mat_row = count1 - 1; mat_row >= 0; --mat_row) {
				double mul_real = input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row* m + count1] *input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1]
								+ input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row* m + count1] *input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1];
					 mul_real /= (input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1] *input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1]
								+ input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1] *input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1]); 

				double mul_imag = input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row* m + count1] *input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1]
								- input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row* m + count1] *input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1];
					 mul_imag /= (input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1] *input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1]
								+ input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1] *input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + count1]); 

				input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * 13 + mat_col] -= (mul_real * input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + mat_col]
																									   - mul_imag * input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + mat_col]); 
				input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * 13 + mat_col] -= (mul_real * input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + mat_col]
																									   + mul_imag * input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + mat_col]);
				__syncthreads(); // APPP

				out_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * 13 + mat_col] -= (mul_real * out_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + mat_col]
																									 - mul_imag * out_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + mat_col]);
				out_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * 13 + mat_col] -= (mul_real * out_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + mat_col]
																									 + mul_imag * out_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + count1 * m + mat_col]);
				__syncthreads(); // APPP
			}

			// wait till all the data is changed
			__syncthreads(); 
		}

		// algorithm: part3 - normalize input data to create matrix inversion
		for (int mat_row = 0; mat_row < 13; ++mat_row) { 
		out_real_temp = out_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * 13 + mat_col]; 
		out_imag_temp = out_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * 13 + mat_col]; 
		out_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * 13 + mat_col] = (out_real_temp * input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * m + mat_row]
																							+ out_imag_temp * input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * m + mat_row])
					 / (input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * m + mat_row] * input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * m + mat_row]
					  + input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * m + mat_row] * input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * m + mat_row]);
		
		out_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * 13 + mat_col] = (out_imag_temp * input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * m + mat_row]
																							- out_real_temp * input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * m + mat_row])
					 / (input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * m + mat_row] * input_reald[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * m + mat_row]
					  + input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * m + mat_row] * input_imagd[thr_batch*data_num*X_size*169+blockIdx.x * 169 + mat_row * m + mat_row]);
		}
		__syncthreads();  // AP


// ======================================================================================================================================









	// ============================================================================
	// initialize shared memroy to zero
		W_reald_shr[thr_batch * data_num * X_size * 13 + blockIdx.x * 13 + mat_col] = 0.0; // ch ...................
		W_imagd_shr[thr_batch * data_num * X_size * 13 + blockIdx.x * 13 + mat_col] = 0.0; // ch ...................

	__syncthreads(); 



	// APC algorithm pqrt4: inv(C+R) * s 
		for (int count1 = 0; count1 < 13; ++count1) {     // count1 < 13
			W_reald_shr[thr_batch * data_num * X_size * 13 + blockIdx.x * 13 + mat_col] += out_reald[thr_batch*data_num*X_size*169+ blockIdx.x * 169 + mat_col * 13 + count1] * s_reald[count1]; 
			W_reald_shr[thr_batch * data_num * X_size * 13 + blockIdx.x * 13 + mat_col] -= out_imagd[thr_batch*data_num*X_size*169+ blockIdx.x * 169 + mat_col * 13 + count1] * s_imagd[count1]; 

			W_imagd_shr[thr_batch * data_num * X_size * 13 + blockIdx.x * 13 + mat_col] += out_reald[thr_batch*data_num*X_size*169+ blockIdx.x * 169 + mat_col * 13 + count1] * s_imagd[count1]; 
			W_imagd_shr[thr_batch * data_num * X_size * 13 + blockIdx.x * 13 + mat_col] += out_imagd[thr_batch*data_num*X_size*169+ blockIdx.x * 169 + mat_col * 13 + count1] * s_reald[count1]; 

		//	W_reald_shr[blockIdx.x * 13 + threadIdx.x] += out_real_shr[threadIdx.x * 13 + count1] * s_reald[count1]; 
		//	W_reald_shr[blockIdx.x * 13 + threadIdx.x] -= out_imag_shr[threadIdx.x * 13 + count1] * s_imagd[count1]; 

		//	W_imagd_shr[blockIdx.x * 13 + threadIdx.x] += out_real_shr[threadIdx.x * 13 + count1] * s_imagd[count1]; 
		//	W_imagd_shr[blockIdx.x * 13 + threadIdx.x] += out_imag_shr[threadIdx.x * 13 + count1] * s_reald[count1]; 
		}




		// modifying ==========================
		//for (int thr_row = 0; thr_row < 13; ++thr_row) {
			//test_reald[thr_batch * data_num * X_size * 169 + blockIdx.x * 169 + thr_row * 13 + thr_col] = out_imagd[thr_batch * data_num * X_size * 169 + blockIdx.x * 169 + thr_row * 13 + thr_col];
			//__syncthreads(); 
			//test_imagd[thr_batch * data_num * X_size * 169 + blockIdx.x * 169 + 0 * 13 + thr_col] = W_imagd_shr[thr_batch * data_num * X_size * 169 + blockIdx.x * 169 + thr_row * 13 + thr_col];
			//__syncthreads();
		//}
		// ==================================== 





	__syncthreads(); 
	
	// APC algorithm part5: W = inv(C+R) * s * rho
		W_reald_shr2[thr_batch*data_num*X_size*13+blockIdx.x*13+mat_col]=W_reald_shr[thr_batch*data_num*X_size*13+blockIdx.x*13+mat_col] * rho_reald[thr_batch*data_num*X_size+blockIdx.x+12]
																	   - W_imagd_shr[thr_batch*data_num*X_size*13+blockIdx.x*13+mat_col] * rho_imagd[thr_batch*data_num*X_size+blockIdx.x+12];
		W_imagd_shr2[thr_batch*data_num*X_size*13+blockIdx.x*13+mat_col]=W_reald_shr[thr_batch*data_num*X_size*13+blockIdx.x*13+mat_col] * rho_imagd[thr_batch*data_num*X_size+blockIdx.x+12]
																	   + W_imagd_shr[thr_batch*data_num*X_size*13+blockIdx.x*13+mat_col] * rho_reald[thr_batch*data_num*X_size+blockIdx.x+12];
	__syncthreads(); 







	// APC algorithm part6: W.t() * y_noisy
	if (threadIdx.x % 13 == 0) {			// hard_code: 13 - ch..
		W_reald[thr_batch * data_num * X_size + blockIdx.x] = 0.0; // 0.0
		W_imagd[thr_batch * data_num * X_size + blockIdx.x] = 0.0; // 0.0

		for (int count1 = 0; count1 < 13; ++count1) {        // !!! W_imagd <-> W_reald !!!
			W_imagd[thr_batch*data_num*X_size+blockIdx.x] -= W_reald_shr2[thr_batch*data_num*X_size*13+blockIdx.x*13+count1]*y_n_reald[thr_batch*data_num*X_size+blockIdx.x+12+12*alpha_indx+count1]
														   - W_imagd_shr2[thr_batch*data_num*X_size*13+blockIdx.x*13+count1]*y_n_imagd[thr_batch*data_num*X_size+blockIdx.x+12+12*alpha_indx+count1];		
			W_reald[thr_batch*data_num*X_size+blockIdx.x] += W_reald_shr2[thr_batch*data_num*X_size*13+blockIdx.x*13+count1]*y_n_imagd[thr_batch*data_num*X_size+blockIdx.x+12+12*alpha_indx+count1]
														   + W_imagd_shr2[thr_batch*data_num*X_size*13+blockIdx.x*13+count1]*y_n_reald[thr_batch*data_num*X_size+blockIdx.x+12+12*alpha_indx+count1];	
		}

	}



		// modifying ====================================
		/*if (threadIdx.x == 0) {
			test_reald[blockIdx.x] = W_reald[blockIdx.x]; 
			test_imagd[blockIdx.x] = W_imagd[blockIdx.x]; 
		}*/
		// ==============================================



}







/*
__global__ void matrixInversion(double* inputd,		// input data is "inputd"
								double* outputd,	// output data is "outputd"
								const int n,				// number of matrices is n
								const int m				// size of each matrix is m*m
								) {					// we suppose input data is squre matrix




	// define our variables
	__shared__ double out[3 * 3];
	__shared__ double in[3 * 3]; 
	

	// define index of each thread
	long long i;
	i = (blockIdx.z * gridDim.y * gridDim.x) + (blockIdx.y * gridDim.x) + (blockIdx.x);
	i *= blockDim.z * blockDim.y * blockDim.x;
	i += (threadIdx.z * blockDim.y * blockDim.x) + (threadIdx.y * blockDim.x) + (threadIdx.x);


	// data and thread location
	int mat_num = i / (m * m); 
	int mat_ind = i % (m * m); 
	int mat_row = (i % (m * m)) / m; 
	int mat_col = (i % (m * m)) % m; 

	// transfer input data to shared memory
	in[mat_ind] = inputd[i]; 


	// creating eye matrix for gauss jordan elimination
	if (mat_row == mat_col) {	
		out[mat_ind] = 1.0; 
	}
	else {
		out[mat_ind] = 0.0; 
	}

	// Matrix inversion algorithm main body ======================================== 
	// we use Gauss Jordan Algorithm
	// algorithm: part1 - make the input data upper-triangular
	for (int count1 = 0; count1 < m - 1; ++count1) {
		
		// change current row when its pivot is zero
		if (in[count1 * m + count1] == 0) {
			int count2 = count1 + 1; 
			while ((in[count2 * m + count1] == 0) && (count2 < m)) {
				++count2;
			}
			if(mat_row == count1) {
				in[mat_ind] += in[count2 * m + mat_col];
				out[mat_ind] += out[count2 * m + mat_col]; 
			}
			__syncthreads(); 	
		}


		if (mat_row > count1) {
			double mul = in[mat_row * m + count1] / in[count1 * m + count1]; 
			in[mat_ind] -= mul * in[count1 * m + mat_col];
			out[mat_ind] -= mul * out[count1 * m + mat_col];
		}

		// wait till all the data is changed
		__syncthreads(); 
	}


	// algorithm: part2 - make the input data lower-triangular
	for (int count1 = m - 1; count1 > 0; --count1) {
		if (mat_row < count1) {
			double mul = in[mat_row * m + count1] / in[count1 * m + count1]; 
			in[mat_ind] -= mul * in[count1 * m + mat_col];
			out[mat_ind] -= mul * out[count1 * m + mat_col];
		}

		// wait till all the data is changed
		__syncthreads(); 
	}

	// algorithm: part3 - normalize input data to create matrix inversion
	out[mat_ind] /= in[mat_row * m + mat_row]; 
	// ============================================================================


	outputd[i] = out[mat_ind]; 
}
*/






// print matrix
void print_matrix(char* name, double* data, int size, int d_shift) {
	printf("arr : %s\n", name);
	for (int i=0+d_shift; i<size+d_shift; ++i) {
		printf("%d %d : %f\n", i, i-d_shift, data[i]); 
	}
	printf("--------------------\n"); 
}




	

// ======================================================================================================================================
/*	

// Matrix inversion algorithm main body ======================================== 
		// we use Gauss Jordan Algorithm
		// algorithm: part1 - make the input data upper-triangular
		for (int count1 = 0; count1 < m - 1; ++count1) {
		

		__syncthreads();  // AP


			// change current row when its pivot is zero
			if ((input_reald[blockIdx.x * 169 + count1 * m + count1] == 0) && (input_imagd[blockIdx.x * 169 + count1 * m + count1] == 0)) {
				int count2 = count1 + 1; 
				while ((input_reald[blockIdx.x * 169 + count2 * m + count1] == 0) && (input_imagd[blockIdx.x * 169 + count2 * m + count1] == 0) && (count2 < m)) {
					++count2;
				}
				if(mat_row == count1) {
					input_reald[blockIdx.x * 169 + mat_ind] += input_reald[blockIdx.x * 169 + count2 * m + mat_col]; // ch ..
					input_imagd[blockIdx.x * 169 + mat_ind] += input_imagd[blockIdx.x * 169 + count2 * m + mat_col]; // ch ..

					out_reald[blockIdx.x * 169 + mat_ind] += out_reald[blockIdx.x * 169 + count2 * m + mat_col]; 
					out_imagd[blockIdx.x * 169 + mat_ind] += out_imagd[blockIdx.x * 169 + count2 * m + mat_col]; 
				}
				__syncthreads(); 	
			}


		__syncthreads();  // AP



			if (mat_row > count1) {
				double mul_real = input_reald[blockIdx.x * 169 + mat_row * m + count1] * input_reald[blockIdx.x * 169 + count1 * m + count1]
								+ input_imagd[blockIdx.x * 169 + mat_row * m + count1] * input_imagd[blockIdx.x * 169 + count1 * m + count1];
				mul_real /= (input_reald[blockIdx.x * 169 + count1 * m + count1] * input_reald[blockIdx.x * 169 + count1 * m + count1]
						   + input_imagd[blockIdx.x * 169 + count1 * m + count1] * input_imagd[blockIdx.x * 169 + count1 * m + count1]); 

				double mul_imag = input_imagd[blockIdx.x * 169 + mat_row * m + count1] * input_reald[blockIdx.x * 169 + count1 * m + count1]
								- input_reald[blockIdx.x * 169 + mat_row * m + count1] * input_imagd[blockIdx.x * 169 + count1 * m + count1];
				mul_imag /= (input_reald[blockIdx.x * 169 + count1 * m + count1] * input_reald[blockIdx.x * 169 + count1 * m + count1]
						   + input_imagd[blockIdx.x * 169 + count1 * m + count1] * input_imagd[blockIdx.x * 169 + count1 * m + count1]); 

				input_reald[blockIdx.x * 169 + mat_ind] -= (mul_real * input_reald[blockIdx.x * 169 + count1 * m + mat_col]
								   - mul_imag * input_imagd[blockIdx.x * 169 + count1 * m + mat_col]); 
				input_imagd[blockIdx.x * 169 + mat_ind] -= (mul_real * input_imagd[blockIdx.x * 169 + count1 * m + mat_col]
								   + mul_imag * input_reald[blockIdx.x * 169 + count1 * m + mat_col]);

				out_reald[blockIdx.x * 169 + mat_ind] -= (mul_real * out_reald[blockIdx.x * 169 + count1 * m + mat_col]
									- mul_imag * out_imagd[blockIdx.x * 169 + count1 * m + mat_col]);
				out_imagd[blockIdx.x * 169 + mat_ind] -= (mul_real * out_imagd[blockIdx.x * 169 + count1 * m + mat_col]
									+ mul_imag * out_reald[blockIdx.x * 169 + count1 * m + mat_col]);
			}

			// wait till all the data is changed
			__syncthreads(); 
		}


		// algorithm: part2 - make the input data lower-triangular
		for (int count1 = m - 1; count1 > 0; --count1) {
		

		__syncthreads();  // AP

			if (mat_row < count1) {
				double mul_real = input_reald[blockIdx.x * 169 + mat_row * m + count1] * input_reald[blockIdx.x * 169 + count1 * m + count1]
								+ input_imagd[blockIdx.x * 169 + mat_row * m + count1] * input_imagd[blockIdx.x * 169 + count1 * m + count1];
				mul_real /= (input_reald[blockIdx.x * 169 + count1 * m + count1] * input_reald[blockIdx.x * 169 + count1 * m + count1]
						   + input_imagd[blockIdx.x * 169 + count1 * m + count1] * input_imagd[blockIdx.x * 169 + count1 * m + count1]); 

				double mul_imag = input_imagd[blockIdx.x * 169 + mat_row * m + count1] * input_reald[blockIdx.x * 169 + count1 * m + count1]
								- input_reald[blockIdx.x * 169 + mat_row * m + count1] * input_imagd[blockIdx.x * 169 + count1 * m + count1];
				mul_imag /= (input_reald[blockIdx.x * 169 + count1 * m + count1] * input_reald[blockIdx.x * 169 + count1 * m + count1]
						   + input_imagd[blockIdx.x * 169 + count1 * m + count1] * input_imagd[blockIdx.x * 169 + count1 * m + count1]); 

				input_reald[blockIdx.x * 169 + mat_ind] -= (mul_real * input_reald[blockIdx.x * 169 + count1 * m + mat_col]
								   - mul_imag * input_imagd[blockIdx.x * 169 + count1 * m + mat_col]); 
				input_imagd[blockIdx.x * 169 + mat_ind] -= (mul_real * input_imagd[blockIdx.x * 169 + count1 * m + mat_col]
								   + mul_imag * input_reald[blockIdx.x * 169 + count1 * m + mat_col]);

				out_reald[blockIdx.x * 169 + mat_ind] -= (mul_real * out_reald[blockIdx.x * 169 + count1 * m + mat_col]
									- mul_imag * out_imagd[blockIdx.x * 169 + count1 * m + mat_col]);
				out_imagd[blockIdx.x * 169 + mat_ind] -= (mul_real * out_imagd[blockIdx.x * 169 + count1 * m + mat_col]
									+ mul_imag * out_reald[blockIdx.x * 169 + count1 * m + mat_col]);
			}

			// wait till all the data is changed
			__syncthreads(); 
		}

		// algorithm: part3 - normalize input data to create matrix inversion
		out_real_temp = out_reald[blockIdx.x * 169 + mat_ind]; 
		out_imag_temp = out_imagd[blockIdx.x * 169 + mat_ind]; 
		out_reald[blockIdx.x * 169 + mat_ind] = (out_real_temp * input_reald[blockIdx.x * 169 + mat_row * m + mat_row]
							   + out_imag_temp * input_imagd[blockIdx.x * 169 + mat_row * m + mat_row])
							   / (input_reald[blockIdx.x * 169 + mat_row * m + mat_row] * input_reald[blockIdx.x * 169 + mat_row * m + mat_row]
								+ input_imagd[blockIdx.x * 169 + mat_row * m + mat_row] * input_imagd[blockIdx.x * 169 + mat_row * m + mat_row]);
		
		out_imagd[blockIdx.x * 169 + mat_ind] = (out_imag_temp * input_reald[blockIdx.x * 169 + mat_row * m + mat_row]
							   - out_real_temp * input_imagd[blockIdx.x * 169 + mat_row * m + mat_row])
								/ (input_reald[blockIdx.x * 169 + mat_row * m + mat_row] * input_reald[blockIdx.x * 169 + mat_row * m + mat_row]
								 + input_imagd[blockIdx.x * 169 + mat_row * m + mat_row] * input_imagd[blockIdx.x * 169 + mat_row * m + mat_row]);

	
		__syncthreads();  // AP


	} // if (i < 169)

	*/
// ======================================================================================================================================









/*
	// transfer input data to shared memory
	in_real[mat_ind] = input_reald[i]; 
	in_imag[mat_ind] = input_imagd[i]; 

	// creating eye matrix for gauss jordan elimination
	if (mat_row == mat_col) {	
		out_real[mat_ind] = 1.0; 
		out_imag[mat_ind] = 0.0; 
	}
	else {
		out_real[mat_ind] = 0.0; 
		out_imag[mat_ind] = 0.0; 
	}


// ========================================== using shared memory

		// Matrix inversion algorithm main body ======================================== 
		// we use Gauss Jordan Algorithm
		// algorithm: part1 - make the input data upper-triangular
		for (int count1 = 0; count1 < m - 1; ++count1) {
		

			__syncthreads();  // AP

			// change current row when its pivot is zero
			if ((in_real[count1 * m + count1] == 0) && (in_imag[count1 * m + count1] == 0)) {
				int count2 = count1 + 1; 
				while ((in_real[count2 * m + count1] == 0) && (in_imag[count2 * m + count1] == 0) && (count2 < m)) {
					++count2;
				}
				if(mat_row == count1) {
					in_real[mat_ind] += in_real[count2 * m + mat_col]; // ch ..
					in_imag[mat_ind] += in_imag[count2 * m + mat_col]; // ch ..

					out_real[mat_ind] += out_real[count2 * m + mat_col]; 
					out_imag[mat_ind] += out_imag[count2 * m + mat_col]; 
				}
				__syncthreads(); 	
			}


			__syncthreads();  // AP


			if (mat_row > count1) {
				double mul_real = in_real[mat_row * m + count1] * in_real[count1 * m + count1]
								+ in_imag[mat_row * m + count1] * in_imag[count1 * m + count1];
				mul_real /= (in_real[count1 * m + count1] * in_real[count1 * m + count1]
						   + in_imag[count1 * m + count1] * in_imag[count1 * m + count1]); 

				double mul_imag = in_imag[mat_row * m + count1] * in_real[count1 * m + count1]
								- in_real[mat_row * m + count1] * in_imag[count1 * m + count1];
				mul_imag /= (in_real[count1 * m + count1] * in_real[count1 * m + count1]
						   + in_imag[count1 * m + count1] * in_imag[count1 * m + count1]); 

				in_real[mat_ind] -= (mul_real * in_real[count1 * m + mat_col]
								   - mul_imag * in_imag[count1 * m + mat_col]); 
				in_imag[mat_ind] -= (mul_real * in_imag[count1 * m + mat_col]
								   + mul_imag * in_real[count1 * m + mat_col]);

				out_real[mat_ind] -= (mul_real * out_real[count1 * m + mat_col]
									- mul_imag * out_imag[count1 * m + mat_col]);
				out_imag[mat_ind] -= (mul_real * out_imag[count1 * m + mat_col]
									+ mul_imag * out_real[count1 * m + mat_col]);
			}

			// wait till all the data is changed
			__syncthreads(); 
		}


		// algorithm: part2 - make the input data lower-triangular
		for (int count1 = m - 1; count1 > 0; --count1) {
		

			__syncthreads();  // AP

			if (mat_row < count1) {
				double mul_real = in_real[mat_row * m + count1] * in_real[count1 * m + count1]
								+ in_imag[mat_row * m + count1] * in_imag[count1 * m + count1];
				mul_real /= (in_real[count1 * m + count1] * in_real[count1 * m + count1]
						   + in_imag[count1 * m + count1] * in_imag[count1 * m + count1]); 

				double mul_imag = in_imag[mat_row * m + count1] * in_real[count1 * m + count1]
								- in_real[mat_row * m + count1] * in_imag[count1 * m + count1];
				mul_imag /= (in_real[count1 * m + count1] * in_real[count1 * m + count1]
						   + in_imag[count1 * m + count1] * in_imag[count1 * m + count1]); 

				in_real[mat_ind] -= (mul_real * in_real[count1 * m + mat_col]
								   - mul_imag * in_imag[count1 * m + mat_col]); 
				in_imag[mat_ind] -= (mul_real * in_imag[count1 * m + mat_col]
								   + mul_imag * in_real[count1 * m + mat_col]);

				out_real[mat_ind] -= (mul_real * out_real[count1 * m + mat_col]
									- mul_imag * out_imag[count1 * m + mat_col]);
				out_imag[mat_ind] -= (mul_real * out_imag[count1 * m + mat_col]
									+ mul_imag * out_real[count1 * m + mat_col]);
			}

			// wait till all the data is changed
			__syncthreads(); 
		}

		// algorithm: part3 - normalize input data to create matrix inversion
		out_real_shr[mat_ind] = (out_real[mat_ind] * in_real[mat_row * m + mat_row]
							   + out_imag[mat_ind] * in_imag[mat_row * m + mat_row])
							   / (in_real[mat_row * m + mat_row] * in_real[mat_row * m + mat_row]
								+ in_imag[mat_row * m + mat_row] * in_imag[mat_row * m + mat_row]);
		
		out_imag_shr[mat_ind] = (out_imag[mat_ind] * in_real[mat_row * m + mat_row]
							   - out_real[mat_ind] * in_imag[mat_row * m + mat_row])
								/ (in_real[mat_row * m + mat_row] * in_real[mat_row * m + mat_row]
								 + in_imag[mat_row * m + mat_row] * in_imag[mat_row * m + mat_row]);


		__syncthreads();  // AP


	} // if (i < 169)


// ================================================================================================

*/
